#include <opencv2/opencv.hpp>
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <filesystem>  // C++17

namespace fs = std::filesystem;

double compute_blur(const cv::Mat& img) {
    cv::Mat gray, lap;
    cv::cvtColor(img, gray, cv::COLOR_BGR2GRAY);
    cv::Laplacian(gray, lap, CV_64F);
    cv::Scalar mean, stddev;
    cv::meanStdDev(lap, mean, stddev);
    return stddev.val[0] * stddev.val[0];
}

int main() {
    std::string folder_path = "images";
    std::string csv_filename = "blur_results.csv";
    std::ofstream csv_file(csv_filename);
    if (!csv_file.is_open()) {
        std::cerr << "Failed to open CSV file for writing.\n";
        return -1;
    }

    csv_file << "Filename,BlurScore,Blurry\n";

    for (const auto& entry : fs::directory_iterator(folder_path)) {
        if (!entry.is_regular_file()) continue;

        std::string file_path = entry.path().string();
        std::string file_name = entry.path().filename().string();

        cv::Mat img = cv::imread(file_path);
        if (img.empty()) {
            std::cerr << "Failed to load: " << file_path << "\n";
            continue;
        }

        double score = compute_blur(img);
        bool is_blurry = score < 100.0;

        std::cout << file_name << ": Score = " << score
                  << " => " << (is_blurry ? "Blurry" : "Not Blurry") << "\n";

        csv_file << file_name << "," << score << "," << (is_blurry ? "Yes" : "No") << "\n";
    }

    csv_file.close();
    std::cout << "Results saved to: " << csv_filename << "\n";

    return 0;
}
